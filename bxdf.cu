#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define GLM_FORCE_CUDA
#include <glm.hpp>

#include "bxdf.h"
#include "rand.h"

using namespace std;
using namespace crs;

__host__ int crs::BxdfTable::getBxdfIdbyName(std::string bxdfname) {
	unsigned int i = 0;

	for (i = 0; i < size; i++) {
		if (bxdfname == toc[i].bxdf_name) return i;
	}

	return i;
}

__device__ void crs::bxdf_NOHIT(Bxdf *b, HitRecord *r, PixelBuffer *p, int pathlength) {
	// calculate and accumulate color
	float t = 0.5 * (r->in.direction.y + 1.0f);
	p->color += ((1.0f - t) * (vec3(1.0f, 1.0f, 1.0f)) + (t * b->ka)) / (float)pathlength;
}

__device__ void crs::bxdf_NORMAL(HitRecord *r, PixelBuffer *p, int pathlength) {
	// accumulate color
	vec3 C = (0.5f * (r->normal + vec3(1.0f, 1.0f, 1.0f))) / (float)pathlength;
	p->color += C;
}

__device__ void crs::bxdf_BSDF(Bxdf *b, HitRecord *r, PixelBuffer *p, int pathlength, unsigned int seed, unsigned int id) {
	// accumulate the color, absorb 50%
	float absorption = 0.5f;
	p->color += (b->ka * r->in.attenuation * absorption) / (float)pathlength;

	// generate a point within a unit sphere and transform according to location and normal
	hiprandState rngState;
	hiprand_init(crs::WangHash(seed)+id, 0, 0, &rngState);
	vec3 t = crs::RandUniformInSphere(&rngState);
	vec3 target = t + r->normal*0.5f + r->location;

	// construct the new ray
	r->in = Ray();
	r->in.origin = r->location;
	r->in.attenuation = glm::length( t );
	r->in.direction = glm::normalize(target - r->location);

	// reset the bxdf
	r->bxdf = NOHIT;
}

__device__ void crs::bxdf_BRDF(Bxdf *b, HitRecord *r, PixelBuffer *p, int pathlength, unsigned int seed, unsigned int id) {
}

__device__ void crs::bxdf_BTDF(Bxdf *b, HitRecord *r, PixelBuffer *p, int pathlength, unsigned int seed, unsigned int id) {
}

__device__ void crs::bxdf_BSSDF(Bxdf *b, HitRecord *r, PixelBuffer *p, int pathlength, unsigned int seed, unsigned int id) {
}

__device__ void crs::bxdf_CONSTANT(Bxdf *b, HitRecord *r, PixelBuffer *p, int pathlength) {
	p->color += b->ka / (float)pathlength;
}

__device__ void crs::evaluateBxdf(Bxdf *bxdfList, HitRecord *r, PixelBuffer *p, int pathlength, unsigned int seed, unsigned int id) {
	
	// early exit if the hitrecord is marked as terminated
	if(r->is_terminated){
		//r->reset();
		return;
	}

	// retrieve the bxdf at intersection
	int bid = r->bxdf;

	switch (bxdfList[bid].type) {
	case crs::NOHIT:
		bxdf_NOHIT(&bxdfList[bid], r, p, pathlength);
		break;
	case crs::NORMAL:
		bxdf_NORMAL(r, p, pathlength);
		break;
	case crs::BSDF:
		bxdf_BSDF(&bxdfList[bid], r, p, pathlength, seed, id);
		break;
	case crs::BRDF:
		bxdf_BRDF(&bxdfList[bid], r, p, pathlength, seed, id);
		break;
	case crs::BTDF:
		bxdf_BTDF(&bxdfList[bid], r, p, pathlength, seed, id);
		break;
	case crs::BSSDF:
		bxdf_BSSDF(&bxdfList[bid], r, p, pathlength, seed, id);
		break;
	case crs::CONSTANT:
		bxdf_CONSTANT(&bxdfList[bid], r, p, pathlength);
		break;
	default:
		// no valid bxdf assigned
		bxdf_NOHIT(&bxdfList[bid], r, p, pathlength);
		break;
	}

	// mark another bounce
	r->pathcounter++;

	// increase sample count if we reached the end of our path
	if(r->pathcounter >= pathlength){
		// reset the hit record for the next sample
		r->reset();
		p->samples += 1;
	}
}

__global__ void crs::KERNEL_BXDF(Bxdf *bxdfList, HitRecord *hitRecords, PixelBuffer *pixelBuffer, int width, int height, int pathlength, unsigned int seed) {
	unsigned long blockId = blockIdx.x + blockIdx.y * gridDim.x;
	unsigned long threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	if (threadId >= width * height) return;
	
	// Evaluate
	evaluateBxdf(bxdfList, &hitRecords[threadId], &pixelBuffer[threadId], pathlength, seed, threadId);
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define GLM_FORCE_CUDA
#include <glm.hpp>
#include <glm/gtc/matrix_transform.hpp>

#include "camera.h"
#include "rand.h"

using namespace std;
using namespace crs;

__host__ __device__ crs::Camera::Camera(){
	width = 100;
	height = 100;
	resolution = 1.0f;
	focusplane = 1.0f;

	fov = 90.0f;			// default 90° vertical fov
	aperture = 0.0f;
}

__host__ __device__ crs::Camera::~Camera(){

}

__host__ __device__ void crs::Camera::updateFOV(){
	// given the vertical fov, calculate the distance to the focal plane
	//float theta = fov * ((float)M_PI/180.0f);
	//float half_height = height * 0.5f;

	// TODO
	focusplane = width;//*0.5f;
}

__host__ __device__ void crs::Camera::updateMatrix(){
	matrix = glm::lookAt(position, lookat, up);
}

__device__ void cast(HitRecord *r, Camera *camera, unsigned long id, unsigned int seed){

	// Generate a 2D random coordinate with a uniform distribution
	hiprandState rngState;
	hiprand_init(crs::WangHash(seed) + id, 0, 0, &rngState);
	glm::vec2 xy = crs::RandUniformSquare(&rngState);

	// Calculate direction, starting with pixel indices
	float x_index = fmod( (float)id, (float)camera->width );
	float y_index = id / camera->width;

	float u = (((xy.x - 0.5f) + x_index) - (camera->width * 0.5f)) / camera->resolution;
	float v = (((xy.y - 0.5f) + y_index) - (camera->height * 0.5f)) / camera->resolution;
	float z = camera->focusplane / camera->resolution;

	vec2 disc = (crs::RandUniformDisc(&rngState) * camera->aperture) / camera->resolution;

	vec3 dof;
	dof.x = camera->position.x + disc.x;
	dof.y = camera->position.y + disc.y;
	dof.z = camera->position.z;

	r->wi.origin = dof;

	r->wi.direction.x = u;
	r->wi.direction.y = -v;
	r->wi.direction.z = -z;

	//transform to world cordinates

	vec3 n = glm::normalize(r->wi.origin + r->wi.direction);
	r->wi.direction = n;

	r->wi.frequency = 0.0f;
	r->wi.length = FLT_MAX;
}

// Generates rays
__global__ void crs::KERNEL_CAST_CAMERA_RAYS(HitRecord *hitrecords, Camera *camera, unsigned int seed){
	unsigned long blockId = blockIdx.x + blockIdx.y * gridDim.x;
	unsigned long threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	if (threadId >= camera->width * camera->height) return;

	// Cast
	cast(&hitrecords[threadId], camera, threadId, seed);
}

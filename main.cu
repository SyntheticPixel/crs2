#include "hip/hip_runtime.h"
// Defs
#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// Includes
#include "main.h"

using namespace std;
using namespace crs;
using namespace rapidjson;

// main
int main(int argc, const char * argv[]){
	Document		dom;
	string			fullpath;
	string			output;
	
	CudaContext		cc;
	
	Camera 			*host_camera;
	Camera			*device_camera;
	
	Sphere			*host_spheres;
	Sphere			*device_spheres;
	
	Triangle		*host_tris;
	Triangle		*device_tris;

	BxdfTable		bxdfTable;
	Bxdf			*host_bxdfs;
	Bxdf			*device_bxdfs;

	float 			gamma_correction;

	unsigned int spherecount = 0;
	unsigned int triscount = 0;
	unsigned int bxdfcount = 0;

#ifdef _WIN32
	if (argc == 2) {
		char	buf[256 + 1];
		string	jsonfile;
		jsonfile = argv[1];
		// Extract the real path
		fullpath = std::string( _fullpath( buf, jsonfile.c_str(), 256) );
		std::string working_directory = fullpath.substr(0, fullpath.find_last_of("\\"));

		// Change working directory to that of the scene file
		const char *wd = working_directory.c_str();
		int d = _chdir(wd);
		if(d != 0){
			cout << " Failed to change the working directory..." << std::endl;
			cout << " ERROR: " << d << " - " << working_directory << std::endl;
			return EXIT_FAILURE;
		}
	}else{
		cout << " ./crs -path-to-json-file" << std::endl;
		return EXIT_FAILURE;
	}
#else
	if (argc == 2) {
		char	buf[256 + 1];
		string	jsonfile;
		jsonfile = argv[1];
		// Extract the real path
		fullpath = std::string(realpath(jsonfile.c_str(), buf));
		std::string working_directory = fullpath.substr(0, fullpath.find_last_of(":/\\"));

		// Change working directory to that of the scene file
		int d = chdir(working_directory.c_str());
		if (d != 0) {
			cout << " Failed to change the working directory..." << std::endl;
			cout << " ERROR: " << working_directory << std::endl;
			return EXIT_FAILURE;
		}
	}
	else {
		cout << " Use : ./crs -path-to-json-file" << std::endl;
		return EXIT_FAILURE;
}
#endif

	// Print out some stats
	cc.GetDeviceProps();
	if(cc.state == CRS_NO_ERROR){
		cc.PrintCudaContext();
	}else{
		cc.EvaluateState();
		return EXIT_SUCCESS;
	}

	// Read the json file
	cout << " Opening scene description: " << fullpath << std::endl;
	ifstream ifs(fullpath);
	IStreamWrapper isw(ifs);
	dom.ParseStream(isw);

	if(dom.HasParseError()){
		ParseErrorCode e = dom.GetParseError();
		cout << " Error parsing json file, error #" << e << std::endl;
		return EXIT_FAILURE;
	}

	if(dom.HasMember("rendersettings")){
		cout << " Rendersettings found..." << std::endl;
		// read the rendersettings
		Value *setting;
		setting = Pointer("/rendersettings/output").Get(dom);
		output = setting->GetString();
		
		setting = Pointer("/rendersettings/width").Get(dom);
		cc.width = setting->GetInt();
		cout << " Image width : " << cc.width << std::endl;
		
		setting = Pointer("/rendersettings/height").Get(dom);
		cc.height = setting->GetInt();
		cout << " Image height : " << cc.height << std::endl;
		
		setting = Pointer("/rendersettings/samples").Get(dom);
		cc.samples = setting->GetInt();
		cout << " Render samples : " << cc.samples << std::endl;
		
		setting = Pointer("/rendersettings/depth").Get(dom);
		cc.depth = setting->GetInt();
		cout << " Path depth : " << cc.depth << std::endl;
		
		setting = Pointer("/rendersettings/gamma").Get(dom);
		gamma_correction = setting->GetFloat();
		cout << " Gamma correction : " << gamma_correction << std::endl;
	}else{
		cout << " Error parsing scene file, no rendersettings found!" << std::endl;
		return EXIT_FAILURE;
	}

	if(dom.HasMember("camera")){
		cout << " Camera found..." << std::endl;

		// read the camera settings
		host_camera = new Camera;

		host_camera->resolution_x = (float)cc.width;
		host_camera->resolution_y = (float)cc.height;
		
		Value *setting;

		setting = Pointer("/camera/position/0").Get(dom);
		host_camera->position.x = setting->GetFloat();
		setting = Pointer("/camera/position/1").Get(dom);
		host_camera->position.y = setting->GetFloat();
		setting = Pointer("/camera/position/2").Get(dom);
		host_camera->position.z = setting->GetFloat();

		setting = Pointer("/camera/lookat/0").Get(dom);
		host_camera->lookat.x = setting->GetFloat();
		setting = Pointer("/camera/lookat/1").Get(dom);
		host_camera->lookat.y = setting->GetFloat();
		setting = Pointer("/camera/lookat/2").Get(dom);
		host_camera->lookat.z = setting->GetFloat();

		setting = Pointer("/camera/up/0").Get(dom);
		host_camera->up.x = setting->GetFloat();
		setting = Pointer("/camera/up/1").Get(dom);
		host_camera->up.y = setting->GetFloat();
		setting = Pointer("/camera/up/2").Get(dom);
		host_camera->up.z = setting->GetFloat();

		setting = Pointer("/camera/field_of_view").Get(dom);
		host_camera->fov = setting->GetFloat();

		setting = Pointer("/camera/focus_distance").Get(dom);
		host_camera->focus_distance = setting->GetFloat();

		setting = Pointer("/camera/aperture_radius").Get(dom);
		host_camera->aperture_radius = setting->GetFloat();

		// make the camera current
		host_camera->update();

	}else{
		cout << " Error parsing scene file, no camera found!" << std::endl;
		return EXIT_FAILURE;
	}

	if(dom.HasMember("bxdfs")){
		Value *setting;
		setting = Pointer("/bxdfs").Get(dom);
		assert(setting->IsArray());
		cout << " " << setting->Size() << " Bxdf(s) found..." << std::endl;

		bxdfTable.size = setting->Size();
		bxdfTable.toc = new BxdfTocEntry[bxdfTable.size];
		host_bxdfs = new Bxdf[bxdfTable.size];
		bxdfcount = bxdfTable.size;

		// fill in the data
		int i = 0;
		for (Value::ConstValueIterator itr = setting->Begin(); itr != setting->End(); ++itr) {
			
			BxdfTocEntry e;
			Bxdf b;

			const Value& name = (*itr)["name"];
			e.bxdf_name = name.GetString();
			e.id = i;
			bxdfTable.toc[i] = e;

			const Value& type = (*itr)["type"];
			const char *temp = type.GetString();

			if (strcmp(temp, "NOHIT") == 0) b.type = crs::NOHIT;
			if (strcmp(temp, "NORMAL") == 0) b.type = crs::NORMAL;
			if (strcmp(temp, "LAMBERT") == 0) b.type = crs::LAMBERT;
			if (strcmp(temp, "OREN_NAYAR") == 0) b.type = crs::OREN_NAYAR;
			if (strcmp(temp, "CONDUCTOR") == 0) b.type = crs::CONDUCTOR;
			if (strcmp(temp, "MICRO_FACET") == 0) b.type = crs::MICRO_FACET;
			if (strcmp(temp, "DIELECTRIC") == 0) b.type = crs::DIELECTRIC;
			if (strcmp(temp, "EMISSION") == 0) b.type = crs::EMISSION;
			if (strcmp(temp, "SUBSURFACE") == 0) b.type = crs::SUBSURFACE;
			if (strcmp(temp, "CONSTANT") == 0) b.type = crs::CONSTANT;
			if (strcmp(temp, "SIMPLE_SKY") == 0) b.type = crs::SIMPLE_SKY;

			const Value& kd = (*itr)["diffuse"];
			b.diffuse.x = kd[0].GetFloat();
			b.diffuse.y = kd[1].GetFloat();
			b.diffuse.z = kd[2].GetFloat();

			const Value& rpt = (*itr)["roughness"];
			b.roughness = rpt.GetFloat();

			const Value& fre = (*itr)["fresnel"];
			b.fresnel = fre.GetFloat();

			const Value& ior = (*itr)["refraction"];
			b.refraction = ior.GetFloat();

			host_bxdfs[i] = b;

			//cout << " Bxdf " << e.id << ", name: " << e.bxdf_name << ", type (id): " << temp << " (" << b.type << ")" << std::endl;
			//cout << " ->Kd: " << b.kd.x << "f, " << b.kd.y << "f, " << b.kd.z << "f" << std::endl;
			//cout << " ->Sh: " << b.sh << "f" << std::endl;

			i++;
		}

	}else{
		cout << " Error parsing scene file, no bxdfs found!" << std::endl;
		return EXIT_FAILURE;
	}

	if(dom.HasMember("spheres")){
		Value *setting;
		setting = Pointer("/spheres").Get(dom);
		assert(setting->IsArray());
		spherecount = setting->Size();
		cout << " " << spherecount << " Sphere(s) found..." << std::endl;

		// assign the memory
		host_spheres = new Sphere[spherecount];
		
		// fill in the data
		int i = 0;
		for (Value::ConstValueIterator itr = setting->Begin(); itr != setting->End(); ++itr){
			Sphere s;

			const Value& c = (*itr)["center"];
			s.center.x = c[0].GetFloat();
			s.center.y = c[1].GetFloat();
			s.center.z = c[2].GetFloat();

			const Value& r = (*itr)["radius"];
			s.radius = r.GetFloat();

			const Value& b = (*itr)["bxdf"];
			std::string name = b.GetString();
			s.bxdf = bxdfTable.getBxdfIdbyName(name);

			host_spheres[i] = s;

			//cout << " Sphere " << i << ", bxdf id:" << s.bxdf << std::endl;

			i++;
		}

	}else{
		cout << " WARNING: no spheres found!" << std::endl;
		//return EXIT_FAILURE;
	}

	if(dom.HasMember("triangles")){
		Value *setting;
		setting = Pointer("/triangles").Get(dom);
		assert(setting->IsArray());
		triscount = setting->Size();
		cout << " " << triscount << " Triangle(s) found..." << std::endl;

		// assign the memory
		host_tris = new Triangle[triscount];

		// fill in the data
		int i = 0;
		for (Value::ConstValueIterator itr = setting->Begin(); itr != setting->End(); ++itr){
			Triangle t;

			const Value& c0 = (*itr)["v0"];
			t.v0.x = c0[0].GetFloat();
			t.v0.y = c0[1].GetFloat();
			t.v0.z = c0[2].GetFloat();

			const Value& c1 = (*itr)["v1"];
			t.v1.x = c1[0].GetFloat();
			t.v1.y = c1[1].GetFloat();
			t.v1.z = c1[2].GetFloat();

			const Value& c2 = (*itr)["v2"];
			t.v2.x = c2[0].GetFloat();
			t.v2.y = c2[1].GetFloat();
			t.v2.z = c2[2].GetFloat();

			const Value& b = (*itr)["bxdf"];
			std::string name = b.GetString();
			t.bxdf = bxdfTable.getBxdfIdbyName(name);

			host_tris[i] = t;

			//cout << " Triangle " << i << ", bxdf id:" << t.bxdf << std::endl;

			i++;
		}

	}else{
		cout << " WARNING: no triangles found!" << std::endl;
		//return EXIT_FAILURE;
	}

	if(spherecount == 0 && triscount == 0){
		cout << " ERROR: no geometry found! Exiting..." << std::endl;
		return EXIT_FAILURE;
	}

	// Device Camera
	hipMalloc((void**)&device_camera, sizeof(crs::Camera));
	hipMemcpy(device_camera, host_camera, sizeof(crs::Camera), hipMemcpyHostToDevice);

	// Device Sphere
	hipMalloc((void**)&device_spheres, sizeof(crs::Sphere)*spherecount);
	hipMemcpy(device_spheres, host_spheres, sizeof(crs::Sphere)*spherecount, hipMemcpyHostToDevice);

	// Device Tris
	hipMalloc((void**)&device_tris, sizeof(crs::Triangle)*triscount);
	hipMemcpy(device_tris, host_tris, sizeof(crs::Triangle)*triscount, hipMemcpyHostToDevice);

	// Device Bxdfs
	hipMalloc((void**)&device_bxdfs, sizeof(crs::Bxdf)*bxdfcount);
	hipMemcpy(device_bxdfs, host_bxdfs, sizeof(crs::Bxdf)*bxdfcount, hipMemcpyHostToDevice);

	// assign the buffers on the host
	cc.SetupHostMemory();
	if(cc.state != CRS_NO_ERROR) cc.EvaluateState();

	// assign the buffers on the device
	cc.SetupDeviceMemory();
	if(cc.state != CRS_NO_ERROR) cc.EvaluateState();

	// --------------------------------------------------------
	// Start rendering
	// --------------------------------------------------------
	size_t start, end, elapsed;

	cc.dimension = k2D;
	cc.CalculateLaunchParamaters();

	// Prepare buffers
	crs::KERNEL_INIT <<<cc.gridSize, cc.blockSize>>>(cc.device_hitRecords, cc.device_pixels, cc.width, cc.height);
	hipDeviceSynchronize();

	start = clock();
	// for each sample
	for (int i = 0; i < cc.samples; i++){

		crs::KERNEL_CAST_CAMERA_RAYS <<<cc.gridSize, cc.blockSize>>>(cc.device_hitRecords, device_camera, clock());
		hipDeviceSynchronize();

		// for each bounce
		for (int j = 0; j < cc.depth; j++){

			crs::KERNEL_TRIANGLEINTERSECT <<<cc.gridSize, cc.blockSize>>>(device_tris, triscount, cc.device_hitRecords, cc.width, cc.height);
			hipDeviceSynchronize();

			crs::KERNEL_SPHEREINTERSECT <<<cc.gridSize, cc.blockSize>>>(device_spheres, spherecount, cc.device_hitRecords, cc.width, cc.height);
			hipDeviceSynchronize();

			crs::KERNEL_BXDF <<<cc.gridSize, cc.blockSize >>>(device_bxdfs, cc.device_hitRecords, cc.device_pixels, cc.width, cc.height, cc.depth, clock());
			hipDeviceSynchronize();
		}

		crs::KERNEL_ACCUMULATE<<<cc.gridSize, cc.blockSize >> >(cc.device_hitRecords, cc.device_pixels, cc.width, cc.height);
		hipDeviceSynchronize();

		cout << "\r Rendered sample " << i + 1 << " of " << cc.samples;
	}
	end = clock();

	elapsed = end - start;
	float secs_elapsed = (float)elapsed / CLOCKS_PER_SEC;
	cout << "\n Rendering done! Finished in: "<< secs_elapsed << " seconds" << std::endl;
	cout << "--------------------------------------------------------------------------------------------- " << std::endl;

	// --------------------------------------------------------
	// End rendering
	// --------------------------------------------------------

	// Copy from device to host
	cc.CopyPixelBufferFromDeviceToHost();
	if(cc.state != CRS_NO_ERROR){
		cc.EvaluateState();
	}else{
		// save the file
		crs::SavePPM(cc.host_pixels, cc.width, cc.height, gamma_correction, output);
		cout << " Output saved to " << "output.ppm" << std::endl;
	}

	// Delete the host Camera
	if(host_camera != NULL){
		delete host_camera;
	}

	// Delete the device Camera
	if(device_camera != NULL){
		hipFree(device_camera);
	}

	// Delete the host spheres
	if(host_spheres != NULL){
		delete[] host_spheres;
	}

	// Delete the host triangles
	if(host_tris != NULL){
		delete[] host_tris;
	}

	// Delete host bxdfs
	if (host_bxdfs != NULL) {
		delete[] host_bxdfs;
	}

	// Delete device bxdfs
	if (device_bxdfs != NULL) {
		hipFree(device_bxdfs);
	}

	// Delete the device spheres
	if(device_spheres != NULL){
		hipFree(device_spheres);
	}

	// Delete the device triangles
	if(device_tris != NULL){
		hipFree(device_tris);
	}

	// Delete all memory
	cc.CleanupDevice();
	if(cc.state != CRS_NO_ERROR) cc.EvaluateState();

	cc.CleanupHost();
	if(cc.state != CRS_NO_ERROR) cc.EvaluateState();

	// reset the CUDA device
	hipDeviceReset();

	cout << "--------------------------------------------------------------------------------------------- " << std::endl;
	cout << " The Crayon Rendering System says goodbye! " << std::endl;
	cout << "--------------------------------------------------------------------------------------------- " << std::endl;

	// get outta here
	return EXIT_SUCCESS;
}
